#include <stdio.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define SRC 0

int vertex_count, edge_count, worklist_size;

struct csrGraph{
    int * offset;
    int * edges;
    int * weights;
    int * dist;
};

__global__ void copyKernel(struct csrGraph * d_graph, int * d_offset, int * d_edges, int * d_weights, int * d_dist){
    d_graph->offset = d_offset;
    d_graph->edges = d_edges;
    d_graph->weights = d_weights;
    d_graph->dist = d_dist;
}

__global__ void copyKernel2(struct csrGraph * d_graph, int ** dist){
    dist[0] = d_graph->dist;
}

__global__ void resetDist(struct csrGraph * d_graph, int vertex_count){
    int global_id = threadIdx.x + blockIdx.x*blockDim.x;
    if(global_id==SRC)d_graph->dist[global_id]=0;
    else if(global_id<vertex_count)d_graph->dist[global_id]=1e9;
}

struct csrGraph * initialize(int vertex_count, int edge_count){
    struct csrGraph *graph = (struct csrGraph*)malloc(sizeof(struct csrGraph));
    graph->offset = (int*)malloc((vertex_count+1)*sizeof(int));
    graph->edges = (int*)malloc((edge_count)*sizeof(int));
    graph->weights = (int*)malloc((edge_count)*sizeof(int));
    graph->dist = (int*)malloc((vertex_count)*sizeof(int));
    for(int i=0;i<vertex_count;i+=1)
        graph->dist[i] = 1e9;
    graph->dist[SRC]=0;
    return graph;
}

struct csrGraph * getInput(){
    scanf("%d %d", &vertex_count, &edge_count);

    struct csrGraph *graph = initialize(vertex_count, edge_count);

    for(int i=0;i<vertex_count+1;i+=1)
        scanf("%d", graph->offset+i);
    for(int i=0;i<edge_count;i+=1)
        scanf("%d", graph->edges+i);
    for(int i=0;i<edge_count;i+=1)
        scanf("%d", graph->weights+i);

    return graph;
}

struct csrGraph * copyGraphToGPU(struct csrGraph * graph){
    struct csrGraph * d_graph;
    hipMalloc(&d_graph, sizeof(struct csrGraph));
    
    int * d_offset, *d_edges, *d_weights, *d_dist;
    hipMalloc(&d_offset, (vertex_count+1)*sizeof(int));
    hipMalloc(&d_edges, (edge_count)*sizeof(int));
    hipMalloc(&d_weights, (edge_count)*sizeof(int));
    hipMalloc(&d_dist, (vertex_count)*sizeof(int));

    hipMemcpy(d_offset, graph->offset, (vertex_count+1)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_edges, graph->edges, (edge_count)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_weights, graph->weights, (edge_count)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_dist, graph->dist, (vertex_count)*sizeof(int), hipMemcpyHostToDevice);
    
    copyKernel<<<1, 1>>>(d_graph, d_offset, d_edges, d_weights, d_dist);
    hipDeviceSynchronize();
    
    return d_graph;
}

void copyGraphToCPU(struct csrGraph * graph, struct csrGraph * d_graph){
    int ** dist;
    hipHostAlloc(&dist, sizeof(int*), 0);
    copyKernel2<<<1, 1>>>(d_graph, dist);
    hipDeviceSynchronize();
    hipMemcpy(graph->dist, dist[0], vertex_count*sizeof(int), hipMemcpyDeviceToHost);
}

void initializeWorklist(int * worklist){
    worklist_size = 5*edge_count;
    worklist = (int*)malloc(2*worklist_size*sizeof(int));
    worklist[0] = SRC;
}

int * copyWorklist(int * worklist){
    int * d_worklist;
    hipMalloc(&d_worklist, 2*worklist_size*sizeof(int));
    hipMemcpy(d_worklist, worklist, 2*worklist_size*sizeof(int), hipMemcpyHostToDevice);
    return d_worklist;
}

__device__ void prefix(int * a){
    int id = threadIdx.x;
    int x=1;
    while(x<1024){
        int to_add =0;
        if(id-x>=0)to_add = a[id-x];
        __syncthreads();
        a[id]+=to_add;
        x*=2;
        __syncthreads();
        __threadfence();
    }
}

__global__ void processKernel(struct csrGraph *graph, int * worklist, int * count, int * count_, int * flag, int worklist_capacity){
    int * in_worklist = (*flag==0)?worklist:(worklist+worklist_capacity);    
    int * out_worklist = (*flag==1)?worklist:(worklist+worklist_capacity);    

    int global_id = threadIdx.x+blockIdx.x*blockDim.x;
    int block_id = threadIdx.x;

    int old_count = (*flag==0)?*count:*count_;

    int updated[9000];
    int updated_list_size=9000;
    int updated_count=0;
    __shared__ int add_count[1024];


    __shared__ int total_work;
    __shared__ int don_box[10*1024];
    __shared__ int idx;
    __shared__ int don_size;

    if(threadIdx.x==0)idx=0, total_work=0, don_size=0;
    __syncthreads();
     

    
    int src_vert, start, end;
    if(global_id<old_count){
        src_vert = in_worklist[global_id];
        start = graph->offset[src_vert];
        end = graph->offset[src_vert+1];
    }
    else{
        src_vert = -1;
        start = 0;
        end = 0;
    }

    atomicAdd(&total_work, end-start);
    __syncthreads();
    
    int ave_work = (1023+total_work)/1024;
    //ave_work-=1;
    //if(ave_work<0)ave_work=0;
    int extra_work=end-start-ave_work;
    if(end-start>ave_work) {

        int st = atomicAdd(&don_size, 2*(end-start-ave_work));
        for(int i=st;i<st+2*(extra_work) and i+1<10*1024;i+=2){
            don_box[i] = src_vert;
            don_box[i+1] = --end;
        }
    }

    __syncthreads();

    if(global_id >= old_count){
        updated_count=0;
    }
    else{
        //int end = graph->offset[src_vert+1];
        
        for(int i=start;i<end;i++){
            int end_vert = graph->edges[i];
            int weight = graph->weights[i];
            
            if(graph->dist[end_vert] > graph->dist[src_vert] + weight){
                graph->dist[end_vert] = graph->dist[src_vert] + weight;
                updated[updated_count++]=end_vert;
                if(updated_count==updated_list_size-1){updated[updated_count++]=src_vert;break;}
            }
        }
        /*for(int i=end;i<graph->offset[src_vert+1];i++){
            int end_vert = graph->edges[i];
            int weight = graph->weights[i];
            
            if(graph->dist[end_vert] > graph->dist[src_vert] + weight){
                graph->dist[end_vert] = graph->dist[src_vert] + weight;
                updated[updated_count++]=end_vert;
            }
        }*/
//        if(updated_count!=0){
//            updated[updated_count++]=in_worklist[global_id];
//        }
//        add_count[block_id] = updated_count;
    }
    /*int old;
    int x;
    do{
        if(updated_count+2>updated_list_size)break;
        old = idx;
        x = atomicCAS(&idx, old, old+2);
        if(x==old and old<don_size and old+1<10*1024){
            int s_v = don_box[old];
            int e_v = graph->edges[don_box[old+1]];
            int w = graph->weights[don_box[old+1]];
            if(graph->dist[e_v]>graph->dist[s_v]+w){
                graph->dist[e_v] = graph->dist[s_v]+w;
                updated[updated_count++]=e_v;
                updated[updated_count++]=s_v;
            }
            x = -1;
        }
    }while(x!=old and old < don_size and old+1<10*1024);*/
    if(idx<don_size and idx<10*1024 and updated_count<updated_list_size-2){
        int don_ind = atomicAdd((int *)&idx, 2);
        while(don_ind<don_size and don_ind+1 < 10*1024 and updated_count<updated_list_size-2){
            int s_v = don_box[don_ind];
            int e_v = graph->edges[don_box[don_ind+1]];
            int w = graph->weights[don_box[don_ind+1]];
            if(graph->dist[e_v]>graph->dist[s_v]+w){
                graph->dist[e_v] = graph->dist[s_v]+w;
                updated[updated_count++]=e_v;
                updated[updated_count++]=s_v;
            }
            if(updated_count>=updated_list_size-2)break;
            don_ind = atomicAdd((int*)&idx, 2);
        }
    }


    /*for(int i=0;i<don_size and i+1 < 10*1024;i+=2){
        if(updated_count>updated_list_size-2)break;
        int s_v = don_box[i];
        if(s_v==src_vert){
            int e_v = graph->edges[don_box[i+1]];
            int w = graph->weights[don_box[i+1]];
            if(graph->dist[e_v]>graph->dist[s_v]+w){
                graph->dist[e_v]=graph->dist[s_v]+w;
                updated[updated_count++]=e_v;
                updated[updated_count++]=s_v;
            }
        }
    }*/



    if(updated_count!=0){
        if(updated_count!=updated_list_size)updated[updated_count++]=in_worklist[global_id];
    }
    add_count[block_id] = updated_count;

    __syncthreads();
    prefix(add_count);

    __shared__ int block_start;
    if(block_id==0)
        block_start= atomicAdd(*flag==0?count_:count, add_count[1023]);
    __syncthreads();
    __threadfence();


    int thread_start = block_start + (block_id==0?0:add_count[block_id-1]);
    for(int i=0;i<updated_count;i+=1){
        out_worklist[thread_start+i] = updated[i];
    }
}

int main(int argc, char* argv[]){
    FILE *output, *time;
    output = fopen(argv[1], "w");
    time = fopen(argv[2], "a");

    struct csrGraph * graph = getInput();

    struct csrGraph * d_graph = copyGraphToGPU(graph);

    int * worklist;
    initializeWorklist(worklist);
    int * d_worklist = copyWorklist(worklist);
    
    int *flag, *count, *count_;
    hipHostAlloc(&flag, sizeof(int), 0);
    hipHostAlloc(&count, sizeof(int), 0);
    hipHostAlloc(&count_, sizeof(int), 0);

    clock_t start, end;
    double tim = 0;
    for(int i=0;i<5;i++){
        start = clock();

        *flag = 0;
        *count = 1;
        *count_ = 0;

        while((*flag==0 && *count!=0) || (*flag==1 && *count_!=0)){
            int c = *flag==0?*count:*count_;
            processKernel<<<(1023+c)/1024, 1024>>>(d_graph, d_worklist, count, count_, flag, worklist_size);
            hipDeviceSynchronize();
            *flag = 1-*flag;
            if(*flag==1)*count=0;
            else *count_=0;
        }

        end = clock();
        printf("\t%d : %f\n",i,((double)(end-start))/CLOCKS_PER_SEC);
        if(i!=0)tim += ((double)(end-start))/CLOCKS_PER_SEC;
        resetDist<<<(1023+vertex_count)/1024, 1024>>>(d_graph, vertex_count);
        d_worklist = copyWorklist(worklist);
    }
    fprintf(time, "%12.6f", tim/4);

    copyGraphToCPU(graph, d_graph);
    hipDeviceSynchronize();

/*
    for(int i=0;i<vertex_count;i++){
        fprintf(output, "%d ", graph->dist[i]);
    } 
*/    
    
}

